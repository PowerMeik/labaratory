#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <sstream>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define ITER_TO_UPDATE 250

#define max_thread 32

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция изменения матрицы
__global__ void step(double* A, double* A_new, size_t size) {
	size_t i = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i == 0 || j == 0 || (i == size - 1) || (j == size - 1))
		return;
	A_new[i * size + j] = 0.25 * (A[i * size + j - 1] + A[(i - 1) * size + j] + A[(i + 1) * size + j] + A[i * size + j + 1]);	
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция востановления границ матрицы
__global__ void restore(double* A, int size){
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	A[i] = 10.0 + i * 10.0 / (size - 1);
	A[i * size] = 10.0 + i * 10.0 / (size - 1);
	A[size - 1 + i * size] = 20.0 + i * 10.0 / (size - 1);
	A[size * (size - 1) + i] = 20.0 + i * 10.0 / (size - 1);
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция разницы матриц
__global__ void subtraction(double* A, double* A_new, size_t size) {
	size_t i = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y;
	A_new[i * size + j] = A[i * size + j] - A_new[i * size + j];
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Значения по умодчанию
double eps = 1E-6;
int size = 512;
int iter_max = 1E6;

int main(int argc, char** argv) {
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Получение значений из командной строки
    for(int arg = 0; arg < argc; arg++){ 
        std::stringstream stream;
        if(strcmp(argv[arg], "-error") == 0){
            stream << argv[arg+1];
            stream >> eps;
        }
        else if(strcmp(argv[arg], "-iter") == 0){
            stream << argv[arg+1];
            stream >> iter_max;
        }
        else if(strcmp(argv[arg], "-size") == 0){
            stream << argv[arg+1];
            stream >> size;
        }
    }

	size_t totalSize = size * size;

	std::cout << "Settings: " << "\n\tMin error: " << eps << "\n\tMax iteration: " << iter_max << "\n\tSize: " << size << "x" << size << std::endl;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Выделения памяти
	double* A = new double[totalSize];

    dim3 block = size / max_thread;

	hipSetDevice(0);

	hipStream_t stream;
	hipStreamCreate(&stream);
	hipGraph_t graph;
	hipGraphExec_t graph_instance;

	double* A_Device, *A_new_Device, *deviceError, *tempStorage = NULL;
	size_t tempStorageSize = 0;

	hipMalloc(&A_Device, sizeof(double) * totalSize);
	hipMalloc(&A_new_Device, sizeof(double) * totalSize);
	hipMalloc(&deviceError, sizeof(double));

	restore<<<block, max_thread>>>(A_Device, size);
	hipMemcpy(A_new_Device, A_Device, sizeof(double) * totalSize, hipMemcpyDeviceToDevice);

	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_new_Device, deviceError, totalSize, stream);
	hipMalloc(&tempStorage, tempStorageSize);
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Создание графа
	hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

	for (size_t i = 0; i < ITER_TO_UPDATE; i += 2) {
		step<<<(block, block), (max_thread, max_thread), 0, stream>>>(A_Device, A_new_Device, size);
		step<<<(block, block), (max_thread, max_thread), 0, stream>>>(A_new_Device, A_Device, size);
	}
	subtraction<<<(block, block), (max_thread, max_thread), 0, stream>>>(A_Device, A_new_Device, size);
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_new_Device, deviceError, totalSize, stream);
	restore<<<block, max_thread, 0, stream>>>(A_new_Device, size);

	hipStreamEndCapture(stream, &graph);
	hipGraphInstantiate(&graph_instance, graph, NULL, NULL, 0);
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Основной цикл
	int iter = 0; 
	double error = 1.0;
	while(iter < iter_max && error > eps) {
		hipGraphLaunch(graph_instance, stream);
		hipMemcpy(&error, deviceError, sizeof(double), hipMemcpyDeviceToHost);
		iter += ITER_TO_UPDATE;
	}
	std::cout << "Result:\n\tIter: " << iter << "\n\tError: " << error << std::endl;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Чистка памяти
	hipFree(A_Device);
	hipFree(A_new_Device);
	hipFree(tempStorage);
	hipGraphDestroy(graph);
	hipStreamDestroy(stream);
	delete[] A;
	return 0;
}