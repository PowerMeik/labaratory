#include "hip/hip_runtime.h"
#include <iostream>
//#include <cstring>
#include <sstream>


#include <hipcub/hipcub.hpp>
#include <cub/block/block_reduce.cuh>

#include "hip/hip_runtime.h"

//#include "sub.cuh"

#define IDX2C(i, j, ld) (((j)*(ld))+(i))


__global__ void heat_equation(double* arr, double* arr2, int N) {
    size_t i = blockIdx.x;
	size_t j = threadIdx.x;
    
	if(!(blockIdx.x == 0 || threadIdx.x == 0))
	{
        arr2[IDX2C(i, j, N)] = (arr[IDX2C(i + 1, j, N)] + arr[IDX2C(i - 1, j, N)] + arr[IDX2C(i, j - 1, N)] + arr[IDX2C(i, j + 1, N)]) * 0.25;
    }
}

__global__ void get_error(double* u, double* u_new, double* out)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx>0)
	{
		out[idx] = fabs(u_new[idx] - u[idx]);
	}
}


void print_array(double *A, int size)
{
    for (int i = 0; i < size; ++i)
    {
        for (int j = 0; j < size; ++j)
        {
            printf("%.6f\t", A[IDX2C(i, j, size)]);
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}


template<typename T>
T extractNumber(char* arr){
    std::stringstream stream;
    stream << arr;
    T result;
    if (!(stream >> result)){
        throw std::invalid_argument("Wrong argument type");
    }
    return result;
}


int main(int argc, char *argv[]) {
    hipSetDevice(2);

    int N = 128;
    int num_of_iter = 1000000;
    double accuracy = 0.000001;

//Получаем параметры из командной строки
    for(int arg = 0; arg < argc; arg++){
        if(std::strcmp(argv[arg], "-eps") == 0){
            accuracy = extractNumber<double>(argv[arg+1]);
            arg++;
        }
        else if(std::strcmp(argv[arg], "-i") == 0){
            num_of_iter = extractNumber<int>(argv[arg+1]);
            arg++;
        }
        else if(std::strcmp(argv[arg], "-s") == 0){
            N = extractNumber<int>(argv[arg+1]);
            arg++;
        }
    }

//Начинаем отсчет времени
    clock_t start = clock();
    
//Объявляем массивы
    int size = N*N*sizeof(double);
    double *arr = (double *)calloc(sizeof(double), size);
    double *arr2 = (double *)calloc(sizeof(double), size);
    double *arr3 = (double *)calloc(sizeof(double), size);

    double delta = 10.0 / (N-1);

//Заполняем массив
    arr[IDX2C(0, 0, N)] = 10.0;
    arr[IDX2C(0, N - 1, N)] = 20.0;
    arr[IDX2C(N - 1, 0, N)] = 20.0;
    arr[IDX2C(N - 1, N - 1, N)] = 30.0;

    for(int i = 1; i < N - 1; i++){
        arr[IDX2C(0, i, N)] = arr[IDX2C(0, i-1, N)] + delta;
        arr[IDX2C(N - 1, i, N)] = arr[IDX2C(N - 1, i-1, N)] + delta;
        arr[IDX2C(i, 0, N)] = arr[IDX2C(i-1, 0, N)] + delta;
        arr[IDX2C(i, N - 1, N)] = arr[IDX2C(i-1, N - 1, N)] + delta;
    }
    memcpy(arr2, arr, size);

//Объявляем и заполняем массивы на cuda
    double* Matrix, *MatrixNew, *Error, *deviceError, *errortemp = 0;

    hipMalloc((&Matrix), size);
    hipMalloc((&MatrixNew), size );
    hipMalloc((&Error), size );  
    hipMalloc((&deviceError), sizeof(double));
    
    hipMemcpy(Matrix, arr, size , hipMemcpyHostToDevice);
    hipMemcpy(MatrixNew, arr2, size , hipMemcpyHostToDevice);
   
    size_t tempsize = 0;
    int k = 0;
    double error = 30;

    hipcub::DeviceReduce::Max(errortemp, tempsize, Error, deviceError, size);
    hipMalloc((&errortemp), tempsize);

    for (; (k < num_of_iter) && (error > accuracy); k++) { 
        //Перезаполняем массив
        heat_equation<<<N-1, N-1>>>(Matrix, MatrixNew, N);
        if(k % 10 == 0){
            //Вычисляем матрицу ошибок
            get_error<<<N, N>>>(Matrix, MatrixNew, Error);
            //Вычисляем максимальную ошибку
            hipcub::DeviceReduce::Max(errortemp, tempsize, Error, deviceError, N*N);
            //Копируем ошибку на устройство
            hipMemcpy(&error, deviceError, sizeof(double), hipMemcpyDeviceToHost);
            //printf("%lf\n", error);
            }
        //Обновляем массив
        double* temp = Matrix;
        Matrix = MatrixNew;
        MatrixNew = temp;
    }

//Заканчиваем считать время
    clock_t end = clock();
    printf("%lf\n", 1.0*(end-start)/CLOCKS_PER_SEC);
    
    printf("%d\n%lf\n", k, error);
    std::cout<<"Error"<<error;

//Очищаем память
    hipFree(Matrix);
    hipFree(MatrixNew);
    free(arr);
    free(arr2);
    return 0;
}