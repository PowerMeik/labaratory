#include "hip/hip_runtime.h"
﻿
#include ""
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <ctime>
#include <string>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "mpi.h"

//значения в углах сетки
#define CORN1 10.0
#define CORN2 20.0
#define CORN3 30.0
#define CORN4 20.0


//функция по подсчету/обновлению ячейк сетке
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//функция получает указатели двух массив. 
//обновляет ячейки первого массива на основе среднего значения четерех ближайших по индексу ячейк из второго массива
//функция являеться global и распоточивает подсчет матрицы на потоки
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void calculationMatrix(double* new_arry, const double* old_array, size_t size, size_t sizePerGpu)
{
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("%d", size);
    if (i != 0 && i != sizePerGpu - 1 && j != 0 && j != size - 1)
    {
        new_arry[i * size + j] = 0.25 * (old_array[i * size + j - 1] + old_array[(i - 1) * size + j] +
            old_array[(i + 1) * size + j] + old_array[i * size + j + 1]);
    }
}  


//функция по вычислению разницы матриц
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//функция получает указатели трех массив. 
//модуль разницы двух первых массивов записывает в третий
//при распоточивание, 1d массивы разбеваются на блоки по 32x32 как 2d массивы
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void getDifferenceMatrix(const double* new_arry, const double* old_array, double* dif)
{   
    /*
    int blockIndex = blockIdx.x + gridDim.y * blockIdx.y;
    int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;


    int arrayIndex = blockIndex * blockDim.x * blockDim.y + threadIndex;
    int GRID_SIZEX = gridDim.x * blockDim.x;
    int GRID_SIZEY = gridDim.y * blockDim.y;
    int i = arrayIndex / GRID_SIZEX;
    int j = arrayIndex % GRID_SIZEX;
    */
    /*
    if (i != 0 && i != GRID_SIZEY - 1 && j != 0 && j != GRID_SIZEX - 1) {
        //printf("%lf = abs(%lf - %lf)\n", dif[i * GRID_SIZEX + j], old_array[i * GRID_SIZEX + j], new_arry[i * GRID_SIZEX + j]);
        dif[i * GRID_SIZEX + j] = std::abs(old_array[i * GRID_SIZEX + j] - new_arry[i * GRID_SIZEX + j]);
    }
    */
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
     dif[idx] = std::abs(old_array[idx] - new_arry[idx]);
}

//основной код
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//получает из коммандной строки значения для размерность сетки, точности обновления сетки, максимального количества итераций
//выделяет память на host и device для сеток
//заполняем сетки начальными значениями
//производим вычисления на GPU
//выводим скорость вычисления, кол. итераций и точнось в консоль
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

    // Получаем значения из коммандной строки
    int GRID_SIZE = std::stoi(argv[2]); // размерность сетки
    double ACC = std::pow(10, -(std::stoi(argv[1]))); // до какой точность обновлять сетку
    int ITER = std::stoi(argv[3]); //  максимальное количество итераций


    int rank, sizeOfTheGroup;
    MPI_Init(&argc, &argv);

    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &sizeOfTheGroup);

    
    hipSetDevice(rank);

    if (rank!=0)
        hipDeviceEnablePeerAccess(rank - 1, 0);
    if (rank!=sizeOfTheGroup-1)
        hipDeviceEnablePeerAccess(rank + 1, 0);

    size_t sizeOfAreaForOneProcess = GRID_SIZE / sizeOfTheGroup;
	size_t startYIdx = sizeOfAreaForOneProcess * rank;

    if (rank!=0)
        hipDeviceEnablePeerAccess(rank - 1, 0);
    if (rank!=sizeOfTheGroup-1)
        hipDeviceEnablePeerAccess(rank + 1, 0);



	


    //выделяем память под 2 сетки размера GRID_SIZExGRID_SIZE
    double* newa, *olda;
    hipHostMalloc(&newa,  sizeof(double) * GRID_SIZE * GRID_SIZE); 
    hipHostMalloc(&olda,  sizeof(double) * GRID_SIZE * GRID_SIZE);

    std::memset(olda, 0, GRID_SIZE * GRID_SIZE * sizeof(double));


    int iter_count = 0; // счетчик итераций
    double error = 1.0; // переменная ошибки
    
    double prop1 = (CORN2 - CORN1) / (GRID_SIZE);
    double prop2 = (CORN3 - CORN1) / (GRID_SIZE);
    double prop3 = (CORN4 - CORN3) / (GRID_SIZE);
    double prop4 = (CORN2 - CORN4) / (GRID_SIZE);

    //записываем значения в углы сеток
    olda[0] = CORN1;
    olda[(GRID_SIZE - 1) * GRID_SIZE] = CORN3;
    olda[GRID_SIZE - 1] = CORN2;
    olda[GRID_SIZE - 1 + GRID_SIZE * (GRID_SIZE - 1)] = CORN4;
    newa[0] = CORN1;
    newa[(GRID_SIZE - 1) * GRID_SIZE] = CORN3;
    newa[GRID_SIZE - 1] = CORN2;
    newa[GRID_SIZE - 1 + GRID_SIZE * (GRID_SIZE - 1)] = CORN4;

    //выделяем память на gpu через cuda для 3 сеток
    double* d_newa,* d_olda, *d_dif;
    hipMalloc((void**)&d_olda, sizeof(double) * GRID_SIZE * GRID_SIZE);
    hipMalloc((void**)&d_newa, sizeof(double) * GRID_SIZE * GRID_SIZE);
    hipMalloc((void**)&d_dif, sizeof(double) * GRID_SIZE * GRID_SIZE);

    //вычисления значений границ сетки
    clock_t beforeinit = clock();
    for (size_t i = 1; i < GRID_SIZE - 1; i++) {
        olda[i] = olda[0] + prop1 * i;
        olda[i * GRID_SIZE] = olda[0] + prop2 * i;
        olda[(GRID_SIZE - 1) * GRID_SIZE + i] = olda[(GRID_SIZE - 1) * GRID_SIZE] + prop3 * i;
        olda[GRID_SIZE * i + GRID_SIZE - 1] = olda[GRID_SIZE * (GRID_SIZE - 1) + GRID_SIZE - 1] + prop4 * i;
        newa[i] = olda[i];
        newa[i * GRID_SIZE] = olda[i * GRID_SIZE];
        newa[(GRID_SIZE - 1) * GRID_SIZE + i] = olda[(GRID_SIZE - 1) * GRID_SIZE + i];
        newa[GRID_SIZE * i + GRID_SIZE - 1] = olda[GRID_SIZE * i + GRID_SIZE - 1];
    }

    if (rank != 0 && rank != sizeOfTheGroup - 1)
	{
		sizeOfAreaForOneProcess += 2;
	}
	else 
	{
		sizeOfAreaForOneProcess += 1;
	}
        
    size_t sizeOfAllocatedMemory = GRID_SIZE * sizeOfAreaForOneProcess;
        
    
    unsigned int threads_x = (GRID_SIZE < 1024) ? GRID_SIZE : 1024;
    unsigned int blocks_y = sizeOfAreaForOneProcess;
    unsigned int blocks_x = GRID_SIZE / threads_x;

    dim3 blockDim1(threads_x, 1);
    dim3 gridDim1(blocks_x, blocks_y);

    hipGraph_t graph;    
    hipGraphCreate(&graph, 0);
    hipGraphExec_t graphExec;
    
    hipStream_t stream, memoryStream;
	hipStreamCreate(&stream);
	hipStreamCreate(&memoryStream);
    
    // размерность блоков и грида 
    dim3 block_dim(32, 32);
    dim3 grid_dim(GRID_SIZE / block_dim.x, GRID_SIZE/ block_dim.y);
    
    size_t offset = (rank != 0) ? GRID_SIZE : 0;

    hipMemset(d_olda, 0, sizeof(double) * sizeOfAllocatedMemory);
	hipMemset(d_newa, 0, sizeof(double) * sizeOfAllocatedMemory);

    // копирование информации с CPU на GPU
    hipMemcpy(d_olda, olda + (startYIdx * GRID_SIZE) - offset, sizeof(double) * sizeOfAllocatedMemory, hipMemcpyHostToDevice); // (CPU) olda -> (GPU) d_olda
    hipMemcpy(d_newa, newa + (startYIdx * GRID_SIZE) - offset, sizeof(double) * sizeOfAllocatedMemory, hipMemcpyHostToDevice); // (CPU) newa -> (GPU) d_newa

    //выделяем память на gpu для переменной, которая будет хранить ошибку на device
    double* max_error = 0;
    hipMalloc((void**)&max_error, sizeof(double));

    std::cout << "Initialization time: " << 1.0 * (clock() - beforeinit) / CLOCKS_PER_SEC << std::endl;
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    //hipGraphNode_t nodes[101];
    for (int i = 0; i < 100; i++) {
        /*
        hipKernelNodeParams params = {0};
        memset(&params, 0, sizeof(hipKernelNodeParams));
        void* paramsargs[2]= {(void**)&d_newa, (void**)&d_olda};
        params.func = (void*)&calculationMatrix;
        params.gridDim = dim3(GRID_SIZE, GRID_SIZE-1, 1);
        params.blockDim = dim3(1, 1, 1);
        params.sharedMemBytes = 0;
        params.kernelParams = paramsargs;
        params.extra = NULL;
        hipGraphAddKernelNode(&nodes[i], graph, NULL, 0, &params);
        */
        calculationMatrix <<<gridDim1, blockDim1, 0, stream>>> (d_newa, d_olda, GRID_SIZE, sizeOfAreaForOneProcess);
        
        // Обмен верхней границей
        if (rank != 0)
		{
            MPI_Sendrecv(d_newa + GRID_SIZE + 1, GRID_SIZE - 2, MPI_DOUBLE, rank - 1, 0, 
                d_newa + 1, GRID_SIZE - 2, MPI_DOUBLE, rank - 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		}
		// Обмен нижней границей
		if (rank != sizeOfTheGroup - 1)
		{
            MPI_Sendrecv(d_newa + (sizeOfAreaForOneProcess - 2) * GRID_SIZE + 1, 
				GRID_SIZE - 2, MPI_DOUBLE, rank + 1, 0,
                d_newa + (sizeOfAreaForOneProcess - 1) * GRID_SIZE + 1, 
				GRID_SIZE - 2, MPI_DOUBLE, rank + 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		}

        if(i < 99){
            double* c = d_olda; 
            d_olda = d_newa;
            d_newa = c;
        }
        /*
        if (i > 0) {
            hipGraphAddDependencies(graph, &nodes[i-1], &nodes[i], 1);
        }
        */
    }
    /*
    hipKernelNodeParams params = {0};
    memset(&params, 0, sizeof(hipKernelNodeParams));
    void* paramsargs[3]= {(void**)&d_newa, (void**)&d_olda, (void**)&d_dif};
    params.func = (void*)&getDifferenceMatrix;
    params.gridDim = grid_dim;
    params.blockDim = block_dim;
    params.sharedMemBytes = 0;
    params.kernelParams = paramsargs;
    params.extra = NULL;
    hipGraphAddKernelNode(&nodes[100], graph, NULL, 0, &params);
    hipGraphAddDependencies(graph, &nodes[99], &nodes[100], 1);
    */
    getDifferenceMatrix <<<blocks_x * blocks_y, threads_x, 0, stream>>> (d_newa, d_olda, d_dif);
    hipStreamEndCapture(stream, &graph);

    hipGraphExec_t instance;
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);


    size_t temp_storage_bytes = 0;
    double* temp_storage = NULL;
    //получаем размер временного буфера для редукции
    hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, d_dif, max_error, GRID_SIZE * sizeOfAreaForOneProcess);

    //выделяем память для буфера
    hipMalloc((void**)&temp_storage, temp_storage_bytes);

    clock_t beforecal = clock();
    
    //алгоритм обновления сетки, работающий пока макс. ошибка не станет меньше или равне нужной точности, или пока количество итерации не превысит максимальное количество.
    while (iter_count < ITER && error > ACC) {
        iter_count+= 100;
        //calculationMatrix <<<GRID_SIZE-1, GRID_SIZE-1>>> (d_newa, d_olda); // расчет матрицы
        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);
        // расчитываем ошибку каждую сотую итерацию
        if(iter_count % 100 == 0){
            //getDifferenceMatrix <<<grid_dim, block_dim >>> (d_newa, d_olda, d_dif); // вычисления разницы матрицы
            hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, d_dif, max_error, sizeOfAllocatedMemory); // нахождение максимума в разнице матрицы
            hipMemcpyAsync(&error, max_error, sizeof(double), hipMemcpyDeviceToHost, stream); // запись ошибки в переменную на host
			
            // Находим максимальную ошибку среди всех и передаём её всем процессам
			MPI_Allreduce((void*)&error,(void*)&error, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
        }

    }
    
    if (rank == 0)
	{
        //вывод времени работы на алгоритма
        std::cout << "Calculation time: " << 1.0 * (clock() - beforecal) / CLOCKS_PER_SEC << std::endl;
        //вывод кол. итерацций и значение ошибки
        std::cout << "Iteration: " << iter_count << " " << "Error: " << error << std::endl;
    }
    //очитска памяти
    hipFree(d_olda);
    hipFree(d_newa);
    hipFree(temp_storage);
    hipFree(olda);
    hipFree(newa);

    MPI_Finalize();
return 0;
}

